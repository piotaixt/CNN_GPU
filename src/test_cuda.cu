#include "hip/hip_runtime.h"
#include "test_cuda.hpp"
#include "cuda_utils.hpp"
#include <iostream>
using namespace std;

const int N = 32;

__global__ void root(float *c) {
  //   c[threadIdx.x + blockDim.x * blockIdx.x] =
  //       sqrt(c[threadIdx.x + blockDim.x * blockIdx.x]);
  c[threadIdx.x] = 0;
}

void print_tab(float *M, int N) {
  std::cout << "Print Tab" << std::endl << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << M[i] << " ";
  }
  std::cout << std::endl;
  std::cout << std::endl;
}

void testCuda() {
  float *c = new float[N];
  float *r = new float[N];
  int cudaCount;
  float *d_c;

  for (int i = 0; i < N; i++) {
    c[i] = i;
  }

  hipDeviceProp_t prop;

  gpuErrchk(hipGetDeviceCount(&cudaCount));
  cout << "Cuda count: " << cudaCount << endl;

  gpuErrchk(hipSetDevice(0));
  gpuErrchk(hipGetDeviceProperties(&prop, 0));

  cout << "maxThreadsPerBlock " << prop.maxThreadsPerBlock << endl;
  cout << "maxGridSize " << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << endl;

  cout << "total glob mem: " << prop.totalGlobalMem << endl;
  cout << "name: " << prop.name << endl;

  print_tab(c, N);

  gpuErrchk(hipMalloc(&d_c, N * sizeof(float)));
  gpuErrchk(hipMemcpy(d_c, c, N * sizeof(float), hipMemcpyHostToDevice));

  //   float theTime = 1;
  //   hipEvent_t myEvent;
  //   hipEvent_t laterEvent;
  //   hipEventCreate(&myEvent);
  //   hipEventCreate(&laterEvent);
  //   hipEventRecord(myEvent, 0);
  //   hipEventSynchronize(myEvent);
  root<<<1, 16>>>(d_c);
gpuErrchk( hipPeekAtLastError() );
gpuErrchk( hipDeviceSynchronize() );

  //   hipEventRecord(laterEvent, 0);
  //   hipEventElapsedTime(&theTime, myEvent, laterEvent);
  //   std::cout << theTime << std::endl;

  gpuErrchk(hipMemcpy(r, d_c, N * sizeof(float), hipMemcpyDeviceToHost));

  print_tab(c, N);
  print_tab(r, N);

  gpuErrchk(hipFree(d_c));
}
